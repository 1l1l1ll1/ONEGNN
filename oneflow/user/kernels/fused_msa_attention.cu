/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/cuda/softmax.cuh"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/maybe.h"
#include "oneflow/core/common/shape.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/framework/user_op_tensor.h"

namespace oneflow {
namespace cuda {
namespace softmax {
template<typename SRC, typename DST = SRC>
struct MSALoadWithBias {
  MSALoadWithBias(const SRC* q, const SRC* m, const SRC* p, const SRC scale, int64_t stride,
                  int64_t row_size)
      : q(q), m(m), p(p), scale(scale), stride(stride), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    Pack<SRC, N> qmk;
    const int64_t offset = (row * row_size + col) / N;
    qmk.storage = *(reinterpret_cast<const PackType<SRC, N>*>(q) + offset);
    Pack<SRC, N> mask;
    const int64_t m_offset = (row / stride * row_size + col) / N;
    mask.storage = *(reinterpret_cast<const PackType<SRC, N>*>(m) + m_offset);
    Pack<SRC, N> pair_bias;
    const int64_t p_offset = (row % stride * row_size + col) / N;
    pair_bias.storage = *(reinterpret_cast<const PackType<SRC, N>*>(p) + p_offset);
#pragma unroll
    for (int i = 0; i < N; ++i) {
      dst[i] = static_cast<DST>(qmk.elem[i]) * static_cast<DST>(scale)
               + static_cast<DST>(mask.elem[i]) + static_cast<DST>(pair_bias.elem[i]);
    }
  }
  const SRC* q;
  const SRC* m;
  const SRC* p;
  const SRC scale;
  int64_t stride;
  int64_t row_size;
};

template<typename SRC, typename DST = SRC>
struct MSALoad {
  MSALoad(const SRC* q, const SRC* m, const SRC scale, int64_t stride, int64_t row_size)
      : q(q), m(m), scale(scale), stride(stride), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    Pack<SRC, N> qmk;
    const int64_t offset = (row * row_size + col) / N;
    qmk.storage = *(reinterpret_cast<const PackType<SRC, N>*>(q) + offset);  // BhS * S2
    Pack<SRC, N> mask;
    const int64_t m_offset = (row / stride * row_size + col) / N;               // B * S, stride=h*S
    mask.storage = *(reinterpret_cast<const PackType<SRC, N>*>(m) + m_offset);  // BhS * S2
#pragma unroll
    for (int i = 0; i < N; ++i) {
      dst[i] =
          static_cast<DST>(qmk.elem[i]) * static_cast<DST>(scale) + static_cast<DST>(mask.elem[i]);
    }
  }
  const SRC* q;
  const SRC* m;
  const SRC scale;
  int64_t stride;
  int64_t row_size;
};

template<typename T, typename ComputeType>
void LaunchMSAWithBiasBroadcastForwardKernel(hipStream_t stream, T* out, const T* qmk,
                                             const T* mask, const T* bias, T scale,
                                             const int64_t stride, const int64_t row_size,
                                             const int64_t rows, const int64_t cols) {
  DirectStore<ComputeType, T> store(out, row_size);
  MSALoadWithBias<T, ComputeType> load(qmk, mask, bias, scale, stride, row_size);
  OF_CUDA_CHECK((DispatchSoftmax<decltype(load), decltype(store), ComputeType>(stream, load, store,
                                                                               rows, cols)));
};

template<typename T, typename ComputeType>
void LaunchMSABroadcastForwardKernel(hipStream_t stream, T* out, const T* qmk, const T* mask,
                                     T scale, const int64_t stride, const int64_t row_size,
                                     const int64_t rows, const int64_t cols) {
  DirectStore<ComputeType, T> store(out, row_size);
  MSALoad<T, ComputeType> load(qmk, mask, scale, stride, row_size);
  OF_CUDA_CHECK((DispatchSoftmax<decltype(load), decltype(store), ComputeType>(stream, load, store,
                                                                               rows, cols)));
};

template<typename SRC, typename DST = SRC>
struct MSAGradStore {
  MSAGradStore(DST* dx, const SRC scale, int64_t row_size)
      : dx(dx), scale(scale), row_size(row_size) {}
  template<int N>
  __device__ void store(const SRC* dout, int64_t row, int64_t col) const {
    Pack<SRC, N> qmk;
    const int64_t offset = (row * row_size + col) / N;
#pragma unroll
    for (int i = 0; i < N; ++i) { qmk.elem[i] = static_cast<DST>(dout[i] * scale); }
    *(reinterpret_cast<PackType<DST, N>*>(dx) + offset) = qmk.storage;
  }
  SRC* dx;
  const SRC scale;
  int64_t row_size;
};

template<typename T, typename ComputeType>
void LaunchMSABroadcastBackwardKernel(hipStream_t stream, T* dx, const T* y, const T* dy, T scale,
                                      const int64_t row_size, const int64_t rows,
                                      const int64_t cols) {
  MSAGradStore<ComputeType, T> store(dx, scale, row_size);
  DirectLoad<T, ComputeType> load_y(y, row_size);
  DirectLoad<T, ComputeType> load_dy(dy, row_size);
  OF_CUDA_CHECK(
      (DispatchSoftmaxGrad<decltype(load_y), decltype(load_dy), decltype(store), ComputeType>(
          stream, load_y, load_dy, store, rows, cols)));
};
}  // namespace softmax

}  // namespace cuda

template<typename T>
class FusedMSAAttentionKernel final : public user_op::OpKernel {
 public:
  FusedMSAAttentionKernel() = default;
  ~FusedMSAAttentionKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* qmk = ctx->Tensor4ArgNameAndIndex("qmk", 0);
    const user_op::Tensor* mask = ctx->Tensor4ArgNameAndIndex("mask", 0);
    const T scale = ctx->Attr<T>("scale");
    const std::string mode = ctx->Attr<std::string>("mode");
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    auto qmk_shape = qmk->shape_view();

    int64_t B = qmk_shape.At(0), h = qmk_shape.At(1), S = qmk_shape.At(2);
    if (ctx->has_input("bias", 0)) {
      const user_op::Tensor* bias = ctx->Tensor4ArgNameAndIndex("bias", 0);
      cuda::softmax::LaunchMSAWithBiasBroadcastForwardKernel<T, T>(
          ctx->stream()->As<ep::CudaStream>()->hip_stream(), out->mut_dptr<T>(), qmk->dptr<T>(),
          mask->dptr<T>(), bias->dptr<T>(), scale, h * S, S, B * h * S, S);
    } else {
      int64_t stride = mode == "col" ? h * S : h;
      int64_t rows = mode == "col" ? B * h * S : h * B;
      cuda::softmax::LaunchMSABroadcastForwardKernel<T, T>(
          ctx->stream()->As<ep::CudaStream>()->hip_stream(), out->mut_dptr<T>(), qmk->dptr<T>(),
          mask->dptr<T>(), scale, stride, S, rows, S);
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_MSA_ATTENTION_KERNEL_GPU(dtype)                 \
  REGISTER_USER_KERNEL("fused_msa_attention")                          \
      .SetCreateFn<FusedMSAAttentionKernel<dtype>>()                   \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_MSA_ATTENTION_KERNEL_GPU(float)
REGISTER_FUSED_MSA_ATTENTION_KERNEL_GPU(double)

template<typename T>
class FusedMSAAttentionGradKernel final : public user_op::OpKernel {
 public:
  FusedMSAAttentionGradKernel() = default;
  ~FusedMSAAttentionGradKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const T scale = ctx->Attr<T>("scale");
    auto y_shape = y->shape_view();

    const int64_t B = y_shape.At(0), h = y_shape.At(1), S = y_shape.At(2);
    cuda::softmax::LaunchMSABroadcastBackwardKernel<T, T>(
        ctx->stream()->As<ep::CudaStream>()->hip_stream(), dx->mut_dptr<T>(), y->dptr<T>(),
        dy->dptr<T>(), scale, S, B * h * S, S);
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_MSA_ATTENTION_GRAD_KERNEL_GPU(dtype)            \
  REGISTER_USER_KERNEL("fused_msa_attention_grad")                     \
      .SetCreateFn<FusedMSAAttentionGradKernel<dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dx", 0) == GetDataType<dtype>::value));

REGISTER_FUSED_MSA_ATTENTION_GRAD_KERNEL_GPU(float)
REGISTER_FUSED_MSA_ATTENTION_GRAD_KERNEL_GPU(double)
}  // namespace oneflow
