#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

struct FusedSinusoidalPositionalEncodeParam {
    const void*  in_ptr;
    float* out_ptr;
    int N;
    int embedding_dim;
    int next_stride;
    int init_offset;
    int stride;
    float downscale_freq_shift;
    float scale;
    int max_period;
};

enum class EncodingPattern {
    SIN_COS,
    COS_SIN,
    INTERLEAVED_SIN_COS,
    INTERLEAVED_COS_SIN
};

template<typename Src>
__global__ void ComputeCosKernel(struct FusedSinusoidalPositionalEncodeParam param) {
    const Src* in_ptr = reinterpret_cast<const Src*>(param.in_ptr);
    float* out_ptr = param.out_ptr;

    for (int offset = threadIdx.x + blockDim.x * blockIdx.x; offset < param.N * param.embedding_dim; offset += blockDim.x * gridDim.x) {
        float position = in_ptr[offset / param.embedding_dim];
        int dim = (offset % param.embedding_dim);
        float exponent = -logf(param.max_period) * dim;
        exponent = exponent / (param.embedding_dim - param.downscale_freq_shift);
        float emb = expf(exponent) * position * param.scale;

        out_ptr[(offset % param.embedding_dim) * param.next_stride + 
            (offset / param.embedding_dim) * param.stride] = cosf(emb);
    }
}

template<typename Src>
__global__ void ComputeSinKernel(struct FusedSinusoidalPositionalEncodeParam param) {
    const Src* in_ptr = reinterpret_cast<const Src*>(param.in_ptr);
    float* out_ptr = param.out_ptr;

    for (int offset = threadIdx.x + blockDim.x * blockIdx.x; offset < param.N * param.embedding_dim; offset += blockDim.x * gridDim.x) {
        float position = in_ptr[offset / param.embedding_dim];
        int dim = (offset % param.embedding_dim);
        float exponent = -logf(param.max_period) * dim;
        exponent = exponent / (param.embedding_dim - param.downscale_freq_shift);
        float emb = expf(exponent) * position * param.scale;

        out_ptr[(offset % param.embedding_dim) * param.next_stride + 
            (offset / param.embedding_dim) * param.stride] = sinf(emb);
    }
}

__global__ void PaddingKernel(float* out_ptr, int N, int embedding_dim) {
    for (int offset = threadIdx.x + blockDim.x * blockIdx.x; offset < N; offset += blockDim.x * gridDim.x) {
        out_ptr[embedding_dim * offset + embedding_dim - 1] = 0.0;
    }
}

class FusedSinusoidalPositionalEncodeKernel final : public user_op::OpKernel, public user_op::CudaGraphSupport {
 public:
  FusedSinusoidalPositionalEncodeKernel() = default;
  ~FusedSinusoidalPositionalEncodeKernel() override = default;


 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    auto* hip_stream = ctx->stream()->As<ep::CudaStream>();
    const user_op::Tensor* positions = ctx->Tensor4ArgNameAndIndex("positions", 0);
    user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("encoded_positions", 0);

    const int N = positions->shape_view().At(0);
    const int embedding_dim = ctx->Attr<int>("embedding_dim");
    const int half_dim = embedding_dim / 2;
    EncodingPattern pattern = static_cast<EncodingPattern>(ctx->Attr<int>("pattern")); //TODO: should be four different types
    const float downscale_freq_shift = ctx->Attr<float>("downscale_freq_shift");
    const float scale = ctx->Attr<float>("scale");
    const int max_period = ctx->Attr<int>("max_period");

    struct FusedSinusoidalPositionalEncodeParam sin_param = {positions->dptr(), 
        reinterpret_cast<float*>(out->mut_dptr()), N, half_dim, 1, 0,
        embedding_dim, downscale_freq_shift, scale, max_period};
    struct FusedSinusoidalPositionalEncodeParam cos_param = {positions->dptr(), 
        reinterpret_cast<float*>(out->mut_dptr()) + half_dim, N, half_dim, 1, half_dim,
        embedding_dim, downscale_freq_shift, scale, max_period};

    const int num_threads = 256;
    const int num_blocks = MIN((N * half_dim + num_threads - 1) / num_threads, 8192);

    if (pattern == EncodingPattern::SIN_COS) {
        // do nothing
    } else if (pattern == EncodingPattern::COS_SIN) {
        cos_param.out_ptr = reinterpret_cast<float*>(out->mut_dptr());
        sin_param.out_ptr = reinterpret_cast<float*>(out->mut_dptr()) + half_dim;
        cos_param.init_offset = 0;
        sin_param.init_offset = half_dim;
    } else if (pattern == EncodingPattern::INTERLEAVED_SIN_COS) {
        sin_param.out_ptr = reinterpret_cast<float*>(out->mut_dptr());
        cos_param.out_ptr = reinterpret_cast<float*>(out->mut_dptr()) + 1;
        sin_param.next_stride = 2;
        cos_param.next_stride = 2;
    } else if (pattern == EncodingPattern::INTERLEAVED_COS_SIN) {
        cos_param.out_ptr = reinterpret_cast<float*>(out->mut_dptr());
        sin_param.out_ptr = reinterpret_cast<float*>(out->mut_dptr()) + 1;
        sin_param.next_stride = 2;
        cos_param.next_stride = 2;
        cos_param.init_offset = half_dim;
        sin_param.init_offset = 0;
    } else {
        //TODO: alarm
    }

    //TODO: Dispatch Src
    ComputeSinKernel<int><<<num_blocks, num_threads>>>(sin_param);
    ComputeCosKernel<int><<<num_blocks, num_threads>>>(cos_param);

    if (embedding_dim % 2 == 1) {
        PaddingKernel<<<(N + 255) / 256, 256>>>(reinterpret_cast<float*>(out->mut_dptr()), N, embedding_dim);
    }
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_FUSED_SINUSOIDAL_POSITIONAL_ENCODE_KERNEL(data_type)               \
  REGISTER_USER_KERNEL("fused_sinusoidal_positional_encode")                            \
      .SetCreateFn<FusedSinusoidalPositionalEncodeKernel>()                            \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("positions", 0) == data_type));

REGISTER_FUSED_SINUSOIDAL_POSITIONAL_ENCODE_KERNEL(DataType::kInt32);
REGISTER_FUSED_SINUSOIDAL_POSITIONAL_ENCODE_KERNEL(DataType::kFloat);

}  // namespace

}  // namespace oneflow
