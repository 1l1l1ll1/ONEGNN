#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/ep/include/primitive/gather.h"
#include "oneflow/core/ep/cpu/primitive/type_seq.h"
#include "oneflow/core/ep/common/primitive/util.h"
#include "oneflow/core/common/nd_index_offset_helper.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
namespace oneflow {
namespace ep {
namespace primitive {
namespace {
template<typename T, typename K, typename IDX, int N>
__global__ void GatherForwardGpu(const IDX batch_dim_size, const IDX out_instance_size,
                                 const IDX indice_instance_size,
                                 NdIndexOffsetHelper<IDX, N> in_helper,
                                 NdIndexOffsetHelper<IDX, N> out_helper, const K* indices,
                                 const T* data, const IDX gather_dim_size, T* output,
                                 const IDX offset) {
  IDX index[N];
  constexpr int index_axis = N - 2;
  CUDA_1D_KERNEL_LOOP_T(IDX, i, batch_dim_size * out_instance_size) {
    out_helper.OffsetToNdIndex(i, index);
    index[index_axis] =
        indices[index[index_axis] + i / out_instance_size * indice_instance_size] - offset;
    T v{};
    if (index[index_axis] >= 0 && index[index_axis] < gather_dim_size) {
      v = data[in_helper.NdIndexToOffset(index)];
    }
    output[i] = v;
  }
}

bool IsSafeUseIndex32(int64_t batch_dim_size, int64_t outer_dim_size, int64_t gather_dim_size,
                      int64_t inner_dim_size, int64_t num_indices) {
  const int64_t in_elem_cnt = batch_dim_size * outer_dim_size * gather_dim_size * inner_dim_size;
  const int64_t out_elem_cnt = batch_dim_size * outer_dim_size * num_indices * inner_dim_size;
  return std::max(out_elem_cnt, in_elem_cnt) < GetMaxVal<int32_t>() / 2;
}

template<typename T, typename K, typename IDX>
void Dispatch2DimNum(ep::Stream* stream, int64_t gather_dim_size, int64_t inner_dim_size,
                     int64_t num_indices, int64_t offset, const K* indices, const T* data,
                     T* output) {
  constexpr int N = 2;
  const int64_t out_instance_size = num_indices * inner_dim_size;
  const int64_t out_elem_cnt = out_instance_size;
  NdIndexOffsetHelper<IDX, N> in_helper(gather_dim_size, inner_dim_size);
  NdIndexOffsetHelper<IDX, N> out_helper(num_indices, inner_dim_size);
  GatherForwardGpu<T, K, IDX, N><<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                                   stream->As<ep::CudaStream>()->hip_stream()>>>(
      /*batch_dim_size*/ 1, out_instance_size, num_indices, in_helper, out_helper, indices, data,
      gather_dim_size, output, offset);
}

template<typename T, typename K, typename IDX>
void Dispatch3DimNum(ep::Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                     int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                     int64_t offset, const K* indices, const T* data, T* output) {
  constexpr int N = 3;
  const int64_t indice_instance_size = num_indices / batch_dim_size;
  const int64_t out_instance_size = outer_dim_size * indice_instance_size * inner_dim_size;
  const int64_t out_elem_cnt = batch_dim_size * out_instance_size;

  const int64_t outest_dim_size = batch_dim_size * outer_dim_size;

  NdIndexOffsetHelper<IDX, N> in_helper(outest_dim_size, gather_dim_size, inner_dim_size);
  NdIndexOffsetHelper<IDX, N> out_helper(outest_dim_size, indice_instance_size, inner_dim_size);
  GatherForwardGpu<T, K, IDX, N><<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                                   stream->As<ep::CudaStream>()->hip_stream()>>>(
      batch_dim_size, out_instance_size, indice_instance_size, in_helper, out_helper, indices, data,
      gather_dim_size, output, offset);
}

template<typename T, typename K, typename IDX>
void Dispatch4DimNum(ep::Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                     int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                     int64_t offset, const K* indices, const T* data, T* output) {
  constexpr int N = 4;
  const int64_t indice_instance_size = num_indices / batch_dim_size;
  const int64_t out_instance_size = outer_dim_size * indice_instance_size * inner_dim_size;
  const int64_t out_elem_cnt = batch_dim_size * out_instance_size;
  NdIndexOffsetHelper<IDX, N> in_helper(batch_dim_size, outer_dim_size, gather_dim_size,
                                        inner_dim_size);
  NdIndexOffsetHelper<IDX, N> out_helper(batch_dim_size, outer_dim_size, indice_instance_size,
                                         inner_dim_size);
  GatherForwardGpu<T, K, IDX, N><<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                                   stream->As<ep::CudaStream>()->hip_stream()>>>(
      batch_dim_size, out_instance_size, indice_instance_size, in_helper, out_helper, indices, data,
      gather_dim_size, output, offset);
}

template<typename T, typename K, typename IDX>
void DispatchNumDims(ep::Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                     int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                     int64_t offset, const K* indices, const void* data, void* output) {
  if (batch_dim_size == 1 && outer_dim_size == 1) {
    Dispatch2DimNum<T, K, IDX>(stream, gather_dim_size, inner_dim_size, num_indices, offset,
                               indices, static_cast<const T*>(data), static_cast<T*>(output));
  } else if (batch_dim_size == 1 || outer_dim_size == 1) {
    Dispatch3DimNum<T, K, IDX>(stream, batch_dim_size, outer_dim_size, gather_dim_size,
                               inner_dim_size, num_indices, offset, indices,
                               static_cast<const T*>(data), static_cast<T*>(output));
  } else {
    Dispatch4DimNum<T, K, IDX>(stream, batch_dim_size, outer_dim_size, gather_dim_size,
                               inner_dim_size, num_indices, offset, indices,
                               static_cast<const T*>(data), static_cast<T*>(output));
  }
}

template<typename K, typename T>
void DispatchIndexSize(ep::Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                       int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                       int64_t offset, const K* indices, const void* data, void* output) {
  if (IsSafeUseIndex32(batch_dim_size, outer_dim_size, gather_dim_size, inner_dim_size,
                       num_indices)) {
    DispatchNumDims<T, K, int32_t>(stream, batch_dim_size, outer_dim_size, gather_dim_size,
                                   inner_dim_size / sizeof(T), num_indices, offset, indices,
                                   static_cast<const T*>(data), static_cast<T*>(output));
  } else {
    DispatchNumDims<T, K, int64_t>(stream, batch_dim_size, outer_dim_size, gather_dim_size,
                                   inner_dim_size / sizeof(T), num_indices, offset, indices,
                                   static_cast<const T*>(data), static_cast<T*>(output));
  }
}

template<typename K, typename T>
bool TryDispatchMovementType(ep::Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                             int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                             int64_t offset, const K* indices, const void* data, void* output) {
  if (reinterpret_cast<uintptr_t>(data) % sizeof(T) == 0
      && reinterpret_cast<uintptr_t>(output) % sizeof(T) == 0 && inner_dim_size % sizeof(T) == 0) {
    DispatchIndexSize<K, T>(stream, batch_dim_size, outer_dim_size, gather_dim_size, inner_dim_size,
                            num_indices, offset, indices, data, output);
    return true;
  } else {
    return false;
  }
}

template<typename K>
void DispatchMovementSize(ep::Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                          int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                          int64_t offset, const K* indices, const void* data, void* output) {
  using Func = bool (*)(ep::Stream * stream, int64_t batch_dim_size, int64_t outer_dim_size,
                        int64_t gather_dim_size, int64_t inner_dim_size, int64_t num_indices,
                        int64_t offset, const K* indices, const void* data, void* output);
  Func funcs[] = {
      TryDispatchMovementType<K, ulonglong2>,  // 16B
      TryDispatchMovementType<K, uint64_t>,    // 8B
      TryDispatchMovementType<K, uint32_t>,    // 4B
      TryDispatchMovementType<K, uint16_t>,    // 2B
      TryDispatchMovementType<K, uint8_t>,     // 1B
  };
  for (size_t i = 0; i < sizeof(funcs) / sizeof(funcs[0]); ++i) {
    if (funcs[i](stream, batch_dim_size, outer_dim_size, gather_dim_size, inner_dim_size,
                 num_indices, offset, indices, data, output)) {
      break;
    }
  }
}

template<typename T, typename K>
void GatherGpuKernel(Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
                     int64_t gather_dim_size, int64_t inner_dim_size, const void* data,
                     int64_t num_indices, const void* indice, int64_t offset, void* output) {
  DispatchMovementSize(stream, batch_dim_size, outer_dim_size, gather_dim_size,
                       inner_dim_size * sizeof(T), num_indices, offset,
                       static_cast<const K*>(indice), data, output);
}

template<typename T, typename K>
class GatherImpl : public Gather {
 public:
  OF_DISALLOW_COPY_AND_MOVE(GatherImpl);
  GatherImpl() = default;
  ~GatherImpl() = default;
  void Launch(Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
              int64_t gather_dim_size, int64_t inner_dim_size, const void* data,
              int64_t num_indices, const void* indice, void* output) override {
    GatherGpuKernel<T, K>(stream, batch_dim_size, outer_dim_size, gather_dim_size, inner_dim_size,
                          data, num_indices, indice, /*offset*/ 0, output);
  }
  void Launch(Stream* stream, int64_t batch_dim_size, int64_t outer_dim_size,
              int64_t gather_dim_size, int64_t inner_dim_size, const void* data,
              int64_t num_indices, const void* indice, int64_t offset, void* output) override {
    GatherGpuKernel<T, K>(stream, batch_dim_size, outer_dim_size, gather_dim_size, inner_dim_size,
                          data, num_indices, indice, offset, output);
  }
};
template<typename T, typename K>
std::unique_ptr<Gather> NewGather() {
  return std::unique_ptr<Gather>(new GatherImpl<T, K>());
}
#define GATHER_DATA_TYPE_SEQ ARITHMETIC_DATA_TYPE_SEQ OF_PP_MAKE_TUPLE_SEQ(bool, DataType::kBool)
#define GATHER_INDEX_TYPE_SEQ INDEX_DATA_TYPE_SEQ OF_PP_MAKE_TUPLE_SEQ(uint32_t, DataType::kUInt32)
class GatherFactoryImpl : public GatherFactory {
 public:
  OF_DISALLOW_COPY_AND_MOVE(GatherFactoryImpl);
  GatherFactoryImpl() = default;
  ~GatherFactoryImpl() override = default;
  std::unique_ptr<Gather> New(DataType data_dtype, DataType indice_type) override {
    std::tuple<DataType, DataType> type_tuple = std::make_tuple(data_dtype, indice_type);
#define MAKE_NEW_GATHER_ENTRY(in_type_pair, indice_type_pair)                             \
  {std::make_tuple(OF_PP_PAIR_SECOND(in_type_pair), OF_PP_PAIR_SECOND(indice_type_pair)), \
   NewGather<OF_PP_PAIR_FIRST(in_type_pair), OF_PP_PAIR_FIRST(indice_type_pair)>},

    static const std::map<std::tuple<DataType, DataType>, std::function<std::unique_ptr<Gather>()>>
        new_gather_handle{OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(
            MAKE_NEW_GATHER_ENTRY, GATHER_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)};

#undef MAKE_NEW_GATHER_ENTRY
    return NewPrimitiveFromHandlers(new_gather_handle, type_tuple);
  }
};
#undef GATHER_INDEX_TYPE_SEQ
#undef GATHER_DATA_TYPE_SEQ
REGISTER_PRIMITIVE_FACTORY(DeviceType::kCUDA, GatherFactory, GatherFactoryImpl);
}  // namespace
}  // namespace primitive
}  // namespace ep
}  // namespace oneflow
